#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <jpeglib.h>
#include <gtkmm.h>
#include "simulation.h"
#include <hip/hip_runtime.h>
//#include "square.cuh"

#include <math.h>

using namespace std;

#define TILE_WIDTH 2

/*matrix multiplication kernels*/

//non shared
__global__ void MatrixMul( float *Md , float *Nd , float *Pd , const int WIDTH )
{

    // calculate thread id
    printf("hello from thread %i\n",threadIdx.x );
    unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
    unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;

    for (int k = 0 ; k<WIDTH ; k++ )
    {
        Pd[row*WIDTH + col]+= Md[row * WIDTH + k ] * Nd[ k * WIDTH + col] ;
        Pd[row*WIDTH + col]+= Md[row * WIDTH + k ] + Nd[ row * WIDTH + k ] ;

    }
}

void try_cuda()
{
    const int WIDTH = 6 ;
    float array1_h[WIDTH][WIDTH] ,array2_h[WIDTH][WIDTH],
            result_array_h[WIDTH][WIDTH] ,M_result_array_h[WIDTH][WIDTH]  ;

    float *array1_d , *array2_d ,*result_array_d  ,*M_result_array_d ; // device array
    int i , j ;
    //input in host array
    for ( i = 0 ; i<WIDTH ; i++ )
    {
        for (j = 0 ; j<WIDTH ; j++ )
        {
            array1_h[i][j] = 1. ;
            array2_h[i][j] = 2. ;
        }
    }

    //create device array hipMalloc ( (void **)&array_name, sizeofmatrixinbytes) ;
    hipMalloc((void **) &array1_d , WIDTH*WIDTH*sizeof (int) ) ;
    hipMalloc((void **) &array2_d , WIDTH*WIDTH*sizeof (int) ) ;

    //copy host array to device array; hipMemcpy ( dest , source , WIDTH , direction )
    hipMemcpy ( array1_d , array1_h , WIDTH*WIDTH*sizeof (int) , hipMemcpyHostToDevice ) ;
    hipMemcpy ( array2_d , array2_h , WIDTH*WIDTH*sizeof (int) , hipMemcpyHostToDevice ) ;

    //allocating memory for resultent device array
    hipMalloc((void **) &result_array_d , WIDTH*WIDTH*sizeof (int) ) ;
    hipMalloc((void **) &M_result_array_d , WIDTH*WIDTH*sizeof (int) ) ;

    //calling kernal
    dim3 dimGrid ( WIDTH/TILE_WIDTH , WIDTH/TILE_WIDTH ,1 ) ;
    dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;


    MatrixMul <<<dimGrid,dimBlock>>> ( array1_d , array2_d ,M_result_array_d , WIDTH) ;


    // all gpu function blocked till kernel is working
    //copy back result_array_d to result_array_h

    hipMemcpy(M_result_array_h , M_result_array_d , WIDTH*WIDTH*sizeof(int) ,
               hipMemcpyDeviceToHost) ;

    //printf the result array
    for ( i = 0 ; i<WIDTH ; i++ )
    {
        for ( j = 0 ; j < WIDTH ; j++ )
        {
            printf ("%f   ",M_result_array_h[i][j] ) ;
        }
        printf ("\n") ;
    }

}


int main(int argc, char* argv[])
{


//    try_cuda();


    Glib::RefPtr<Gtk::Application> app = Gtk::Application::create(argc, argv, "com.kaze.test");

    Simulation sim;

    // The Gui Window is displayed
    return app->run(sim);
}

