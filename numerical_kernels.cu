#include "hip/hip_runtime.h"
////
//// Created by jannik on 22/05/18.
////
//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "numerical_kernels.h"

#define TILE_WIDTH 2

__global__ void heat_kernel( float *data_array_new_device , float *data_array_old_device , const int width )
{

    // calculate thread id
    printf("hello from thread %i\n",threadIdx.x );
    unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
    unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;

    data_array_new_device[row*width + col] = (data_array_old_device[row * width + k ] + 10) % 256;

}



void launch_kernel()
{

    //copy host array to device array; hipMemcpy ( dest , source , WIDTH , direction )
    hipMemcpy ( data_array_new_device , data_array_new_host , height*width*sizeof (float) , hipMemcpyHostToDevice ) ;
    hipMemcpy ( data_array_old_device, data_array_old_host , height*width*sizeof (float) , hipMemcpyHostToDevice ) ;

    //calling kernal
    dim3 dimGrid ( width/TILE_WIDTH , height/TILE_WIDTH ,1 ) ;
    dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;


    heat_kernel <<<dimGrid, dimBlock>>> ( data_array_new_device , data_array_old_device , width) ;

    // copy back resutls
    hipMemcpy(data_array_new_host , data_array_new_device , height*width*sizeof(float), hipMemcpyDeviceToHost) ;

}

void cuda_init()
{


    // populate cuda stuff
    for (int i = 0 ; i<width ; i++ )
    {
        for (int j = 0 ; j<height ; j++ )
        {
            data_array_new_host[i][j] = 1.;
        }
    }

    //create device array hipMalloc ( (void **)&array_name, sizeofmatrixinbytes) ;
    hipMalloc((void **) &data_array_new_device , height*width*sizeof (float) ) ;
}



// TESTING



__global__ void MatrixMul( float *Md , float *Nd , float *Pd , const int WIDTH )
{

    // calculate thread id
    printf("hello from thread %i\n",threadIdx.x );
    unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
    unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;

    for (int k = 0 ; k<WIDTH ; k++ )
    {
        Pd[row*WIDTH + col]+= Md[row * WIDTH + k ] * Nd[ k * WIDTH + col] ;

    }
}

void try_cuda()
{


    const int WIDTH = 100 ;
    float array1_h[WIDTH][WIDTH] ,array2_h[WIDTH][WIDTH],
            result_array_h[WIDTH][WIDTH] ,M_result_array_h[WIDTH][WIDTH]  ;

    float *array1_d , *array2_d ,*result_array_d  ,*M_result_array_d ; // device array
    int i , j ;
    //input in host array
    for ( i = 0 ; i<WIDTH ; i++ )
    {
        for (j = 0 ; j<WIDTH ; j++ )
        {
            array1_h[i][j] = 1. ;
            array2_h[i][j] = 2. ;
        }
    }

    //create device array hipMalloc ( (void **)&array_name, sizeofmatrixinbytes) ;
    hipMalloc((void **) &array1_d , WIDTH*WIDTH*sizeof (int) ) ;
    hipMalloc((void **) &array2_d , WIDTH*WIDTH*sizeof (int) ) ;

    //copy host array to device array; hipMemcpy ( dest , source , WIDTH , direction )
    hipMemcpy ( array1_d , array1_h , WIDTH*WIDTH*sizeof (int) , hipMemcpyHostToDevice ) ;
    hipMemcpy ( array2_d , array2_h , WIDTH*WIDTH*sizeof (int) , hipMemcpyHostToDevice ) ;

    //allocating memory for resultent device array
    hipMalloc((void **) &result_array_d , WIDTH*WIDTH*sizeof (int) ) ;
    hipMalloc((void **) &M_result_array_d , WIDTH*WIDTH*sizeof (int) ) ;

    //calling kernel
    dim3 dimGrid ( WIDTH/TILE_WIDTH , WIDTH/TILE_WIDTH ,1 ) ;
    dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;


    MatrixMul <<<dimGrid,dimBlock>>> ( array1_d , array2_d ,M_result_array_d , WIDTH) ;


    // all gpu function blocked till kernel is working
    //copy back result_array_d to result_array_h

    hipMemcpy(M_result_array_h , M_result_array_d , WIDTH*WIDTH*sizeof(int) ,
               hipMemcpyDeviceToHost) ;

    //printf the result array
    for ( i = 0 ; i<WIDTH ; i++ )
    {
        for ( j = 0 ; j < WIDTH ; j++ )
        {
            printf ("%f   ",M_result_array_h[i][j] ) ;
        }
        printf ("\n") ;
    }

}




void test()
{
    printf("test \n");
}